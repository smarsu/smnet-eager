#include "hip/hip_runtime.h"
// Copyright (c) 2020 smarsu. All Rights Reserved.

#include "core.h"

extern "C" {

struct Deconv2DParams {
  hipdnnTensorDescriptor_t x_desc;
  hipdnnFilterDescriptor_t w_desc;
  hipdnnTensorDescriptor_t y_desc;
  hipdnnTensorDescriptor_t bias_desc;
  hipdnnConvolutionDescriptor_t conv_desc;

  hipdnnConvolutionBwdDataAlgo_t fwd_algo;
  hipdnnConvolutionFwdAlgo_t bwd_data_algo;
  hipdnnConvolutionBwdFilterAlgo_t bwd_filter_algo;

  size_t size = 0;

  float fwd_alpha = 1;
  float fwd_beta = 0;

  float fwd_bias_alpha = 1;
  float fwd_bias_beta = 0;

  float bwd_bias_alpha = 1;
  float bwd_bias_beta = 0;

  float bwd_data_alpha = 1;
  float bwd_data_beta = 0;

  float bwd_filter_alpha = 1;
  float bwd_filter_beta = 0;
};

void DestroyDeconv2DParams(Deconv2DParams *params) {
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->x_desc));
  CALL_CUDNN(hipdnnDestroyFilterDescriptor(params->w_desc));
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->y_desc));
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->bias_desc));
  CALL_CUDNN(hipdnnDestroyConvolutionDescriptor(params->conv_desc));

  delete params;
}

Deconv2DParams *CudnnDeconv2DCreate(hipdnnHandle_t cudnn_handle,
                                    int ni, 
                                    int ci, 
                                    int hi, 
                                    int wi,
                                    int co, 
                                    int hf, 
                                    int wf,
                                    int ho,
                                    int wo,
                                    int hp,
                                    int wp,
                                    int hs,
                                    int ws,
                                    int hd,
                                    int wd) {
  Deconv2DParams *params = new Deconv2DParams;

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->x_desc));
  CALL_CUDNN(hipdnnSetTensor4dDescriptor(params->x_desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        ni,
                                        ci,
                                        hi,
                                        wi));

  CALL_CUDNN(hipdnnCreateFilterDescriptor(&params->w_desc));
  CALL_CUDNN(hipdnnSetFilter4dDescriptor(params->w_desc,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        ci,
                                        co,
                                        hf,
                                        wf));

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->y_desc));
  CALL_CUDNN(hipdnnSetTensor4dDescriptor(params->y_desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        ni,
                                        co,
                                        ho,
                                        wo));

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->bias_desc));
  CALL_CUDNN(hipdnnSetTensor4dDescriptor(params->bias_desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        1,
                                        co,
                                        1,
                                        1));

  CALL_CUDNN(hipdnnCreateConvolutionDescriptor(&params->conv_desc));
  CALL_CUDNN(hipdnnSetConvolution2dDescriptor(params->conv_desc,
                                             hp,
                                             wp,
                                             hs,
                                             ws,
                                             hd,
                                             wd,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));

  size_t size = 0;
  CALL_CUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn_handle,
                                                      params->w_desc,
                                                      params->x_desc,
                                                      params->conv_desc,
                                                      params->y_desc,
                                                      // HIPDNN_CONVOLUTION_BWD_DATA_NO_WORKSPACE,
                                                      HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
                                                      0,
                                                      &params->fwd_algo));
  // params->bwd_data_algo = HIPDNN_CONVOLUTION_BWD_DATA_ALGO_1;
  CALL_CUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn_handle,
                                                          params->w_desc,
                                                          params->x_desc,
                                                          params->conv_desc,
                                                          params->y_desc,
                                                          params->fwd_algo,
                                                          &size));
  params->size = std::max(params->size, size);

  CALL_CUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn_handle,
                                                 params->y_desc,
                                                 params->w_desc,
                                                 params->conv_desc,
                                                 params->x_desc,
                                                //  HIPDNN_CONVOLUTION_FWD_NO_WORKSPACE,
                                                 HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                 0,
                                                 &params->bwd_data_algo));
  // params->fwd_algo = HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM;

  CALL_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle,
                                                     params->y_desc,
                                                     params->w_desc,
                                                     params->conv_desc,
                                                     params->x_desc,
                                                     params->bwd_data_algo,
                                                     &size));
  params->size = std::max(params->size, size);

  CALL_CUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn_handle,
                                                        params->y_desc,
                                                        params->x_desc,
                                                        params->conv_desc,
                                                        params->w_desc,
                                                        // HIPDNN_CONVOLUTION_BWD_FILTER_NO_WORKSPACE,
                                                        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
                                                        0,
                                                        &params->bwd_filter_algo));
  // params->bwd_filter_algo = HIPDNN_CONVOLUTION_BWD_FILTER_ALGO_1;
  CALL_CUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn_handle,
                                                            params->y_desc,
                                                            params->x_desc,
                                                            params->conv_desc,
                                                            params->w_desc,
                                                            params->bwd_filter_algo,
                                                            &size));
  params->size = std::max(params->size, size);

  CudaBuffer::Buffer()->Resize(params->size);
  LOG(INFO) << "Get Deconv2D Params ... " << params->size;

  return params;
}

void CudnnDeconv2DForward(hipdnnHandle_t cudnn_handle,
                          Deconv2DParams *params,
                          float alpha,
                          const void *x,
                          const void *w,
                          float beta,
                          void *y) {
  params->fwd_alpha = alpha;
  params->fwd_beta = beta;

  CALL_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle,
                                          &params->fwd_alpha,
                                          params->w_desc,
                                          w,
                                          params->x_desc,
                                          x,
                                          params->conv_desc,
                                          params->fwd_algo,
                                          CudaBuffer::Buffer()->data(),
                                          params->size,
                                          &params->fwd_beta,
                                          params->y_desc,
                                          y));
}

void CudnnDeconv2DForwardBias(hipdnnHandle_t cudnn_handle,
                              Deconv2DParams *params,
                              float alpha,
                              const float *bias,
                              float beta,
                              float *y) {
  params->fwd_bias_alpha = alpha;
  params->fwd_bias_beta = beta;

  CALL_CUDNN(hipdnnAddTensor(cudnn_handle,
                            &params->fwd_bias_alpha,
                            params->bias_desc,
                            bias,
                            &params->fwd_bias_beta,
                            params->y_desc,
                            y));
}

void CudnnDeconv2DBackwardBias(hipdnnHandle_t cudnn_handle,
                               Deconv2DParams *params,
                               float alpha,
                               const void *dy,
                               float beta,
                               void *db) {
  params->bwd_bias_alpha = alpha;
  params->bwd_bias_beta = beta;

  CALL_CUDNN(hipdnnConvolutionBackwardBias(cudnn_handle,
                                          &params->bwd_bias_alpha,
                                          params->y_desc,
                                          dy,
                                          &params->bwd_bias_beta,
                                          params->bias_desc,
                                          db));
}

void CudnnDeconv2DBackwardData(hipdnnHandle_t cudnn_handle,
                               Deconv2DParams *params,
                               float alpha,
                               const void *w,
                               const void *dy,
                               float beta,
                               void *dx) {
  params->bwd_data_alpha = alpha;
  params->bwd_data_beta = beta;
  
  CALL_CUDNN(hipdnnConvolutionForward(cudnn_handle,
                                     &params->bwd_data_alpha,
                                     params->y_desc,
                                     dy,
                                     params->w_desc,
                                     w,
                                     params->conv_desc,
                                     params->bwd_data_algo,
                                     CudaBuffer::Buffer()->data(),
                                     params->size,
                                     &params->bwd_data_beta,
                                     params->x_desc,
                                     dx));
}

void CudnnDeconv2DBackwardFilter(hipdnnHandle_t cudnn_handle,
                                 Deconv2DParams *params,
                                 float alpha,
                                 const void *x,
                                 const void *dy,
                                 float beta,
                                 void *dw) {
  params->bwd_filter_alpha = alpha;
  params->bwd_filter_beta = beta;

  CALL_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle,
                                            &params->bwd_filter_alpha,
                                            params->y_desc,
                                            dy,
                                            params->x_desc,
                                            x,
                                            params->conv_desc,
                                            params->bwd_filter_algo,
                                            CudaBuffer::Buffer()->data(),
                                            params->size,
                                            &params->bwd_filter_beta,
                                            params->w_desc,
                                            dw));
}

}  // extern "C"
