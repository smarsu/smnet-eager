#include "hip/hip_runtime.h"
// Copyright (c) 2020 smarsu. All Rights Reserved.

#include "core.h"

struct Conv2dParams {
  hipdnnTensorDescriptor_t x_desc;
  hipdnnFilterDescriptor_t w_desc;
  hipdnnTensorDescriptor_t y_desc;
  hipdnnConvolutionDescriptor_t conv_desc;

  hipdnnConvolutionFwdAlgo_t fwd_algo;
  hipdnnConvolutionBwdDataAlgo_t bwd_data_algo;
  hipdnnConvolutionBwdFilterAlgo_t bwd_filter_algo;

  size_t size{0};

  float fwd_alpha{1};
  float fwd_beta{0};

  float bwd_data_alpha{1};
  float bwd_data_beta{0};

  float bwd_filter_alpha{1};
  float bwd_filter_beta{0};

} Conv2dParams_t;

void DestroyParams(Conv2dParams *params) {
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->x_desc));
  CALL_CUDNN(hipdnnDestroyFilterDescriptor(params->w_desc));
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->y_desc));
  CALL_CUDNN(hipdnnDestroyConvolutionDescriptor(params->conv_desc));

  delete params;
}

Conv2dParams *CudnnConv2DCreate(hipdnnHandle_t cudnn_handle,
                                int ni, 
                                int ci, 
                                int hi, 
                                int wi,
                                int co, 
                                int hf, 
                                int wf,
                                int ho,
                                int wo,
                                int hp,
                                int wp,
                                int hs,
                                int ws,
                                int hd,
                                int wd) {
  Conv2dParams *params = new Conv2dParams;

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&param->x_desc));
  CALL_CUDNN(hipdnnSetTensor4dDescriptor(param->x_desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        ni,
                                        ci,
                                        hi,
                                        wi));

  CALL_CUDNN(hipdnnCreateFilterDescriptor(&param->w_desc));
  CALL_CUDNN(hipdnnSetFilter4dDescriptor(param->w_desc,
                                        HIPDNN_DATA_FLOAT,
                                        HIPDNN_TENSOR_NCHW,
                                        co,
                                        ci,
                                        hf,
                                        wf));

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&param->y_desc));
  CALL_CUDNN(hipdnnSetTensor4dDescriptor(param->y_desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        ni,
                                        co,
                                        ho,
                                        wo));
                                      
  CALL_CUDNN(hipdnnCreateConvolutionDescriptor(&params->conv_desc));
  CALL_CUDNN(hipdnnSetConvolution2dDescriptor(params->conv_desc,
                                             hp,
                                             wp,
                                             hs,
                                             ws,
                                             hd,
                                             wd,
                                             HIPDNN_CROSS_CORRELATION,
                                             HIPDNN_DATA_FLOAT));

  CALL_CUDNN(hipdnnGetConvolutionForwardAlgorithm(cudnn_handle,
                                                 params->x_desc,
                                                 params->w_desc,
                                                 params->conv_desc,
                                                 params->y_desc,
                                                 HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST,
                                                 0,
                                                 &params->fwd_algo));

  size_t size = 0;
  CALL_CUDNN(hipdnnGetConvolutionForwardWorkspaceSize(cudnn_handle,
                                                     params->x_desc,
                                                     params->w_desc,
                                                     params->conv_desc,
                                                     params->y_desc,
                                                     params->fwd_algo,
                                                     &size));
  params->size = std::max(params->size, size);

  CALL_CUDNN(hipdnnGetConvolutionBackwardDataAlgorithm(cudnn_handle,
                                                      params->w_desc,
                                                      params->y_desc,
                                                      params->conv_desc
                                                      params->x_desc
                                                      HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST,
                                                      0,
                                                      &params->bwd_data_algo));
  CALL_CUDNN(hipdnnGetConvolutionBackwardDataWorkspaceSize(cudnn_handle,
                                                          params->w_desc,
                                                          params->y_desc,
                                                          params->conv_desc,
                                                          params->x_desc,
                                                          params->bwd_data_algo,
                                                          &size));
  params->size = std::max(params->size, size);

  CALL_CUDNN(hipdnnGetConvolutionBackwardFilterAlgorithm(cudnn_handle,
                                                        params->x_desc,
                                                        params->y_desc,
                                                        params->conv_desc,
                                                        params->w_desc,
                                                        HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST,
                                                        0,
                                                        &params->bwd_filter_algo));
  CALL_CUDNN(hipdnnGetConvolutionBackwardFilterWorkspaceSize(cudnn_handle,
                                                            params->x_desc,
                                                            params->y_desc,
                                                            params->conv_desc,
                                                            params->w_desc,
                                                            params->bwd_filter_algo,
                                                            &size));
  params->size = std::max(params->size, size);

  CudaBuffer::Buffer()->Resize(params->size);
}

void CudnnConv2DForward(hipdnnHandle_t cudnn_handle,
                        Conv2dParams_t params,
                        float alpha,
                        const void *x,
                        const void *w,
                        float beta,
                        void *y) {
  params->fwd_alpha = alpha;
  params->fwd_beta = beta;

  CALL_CUDNN(hipdnnConvolutionForward(cudnn_handle,
                                     &params->fwd_alpha,
                                     params->x_desc,
                                     x
                                     params->w_desc,
                                     w
                                     params->conv_desc,
                                     params->fwd_algp,
                                     CudaBuffer::Buffer(),
                                     params->size,
                                     &params->fwd_beta,
                                     params->y_desc,
                                     y));
}

void CudnnConv2DBackwardData(hipdnnHandle_t cudnn_handle,
                             Conv2dParams_t params,
                             float alpha,
                             const void *w,
                             const void *dy,
                             float beta,
                             void *dx) {
  params->bwd_data_alpha = alpha;
  params->bwd_data_beta = beta;

  CALL_CUDNN(hipdnnConvolutionBackwardData(cudnn_handle,
                                          &params->bwd_data_alpha,
                                          params->w_desc,
                                          w
                                          params->y_desc,
                                          dy,
                                          params->conv_desc,
                                          params->bwd_data_algo,
                                          CudaBuffer::Buffer(),
                                          params->size,
                                          &params->bwd_data_beta,
                                          params->x_desc,
                                          dx));
}

void CudnnConv2DBackwardFilter(hipdnnHandle_t cudnn_handle,
                               Conv2dParams_t params,
                               float alpha,
                               const void *x,
                               const void *dy,
                               void *dw) {
  params->bwd_filter_alpha = alpha;
  params->bwd_filter_beta = beta;

  CALL_CUDNN(hipdnnConvolutionBackwardFilter(cudnn_handle,
                                            &params->bwd_filter_alpha,
                                            params->x_desc,
                                            x
                                            params->y_desc,
                                            dy,
                                            params->conv_desc,
                                            params->bwd_filter_algo,
                                            CudaBuffer::Buffer(),
                                            params->size,
                                            &params->bwd_filter_beta,
                                            params->w_desc,
                                            dw));
}
