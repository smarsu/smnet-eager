// Copyright (c) 2020 smarsu. All Rights Reserved.

#include <vector>
#include <string>
#include <sstream>

#include "core.h"

std::string ToString(int *shape, int ndims) {
  if (ndims == 0) return "";

  std::stringstream ss;
  ss << "[";
  for (int i = 0; i < ndims - 1; ++i) {
    ss << shape[i] << ", ";
  }
  ss << shape[ndims - 1] << "]";
  return ss.str();
}

std::vector<int> Shape2Strides(int *shape, int ndims) {
  std::vector<int> strides(ndims);
  int prod = 1;
  for (int i = ndims - 1; i >= 0; --i) {
    strides[i] = prod;
    prod *= shape[i];
  }

  return strides;
}

hipdnnHandle_t CudnnHandle() {
  static hipdnnHandle_t handle = NULL;
  if (!handle) {
    CALL_CUDNN(hipdnnCreate(&handle)) << " Create cudnn handle failed."; 
  }
  return handle;
}

void *CudaMalloc(size_t size) {
  void *ptr = NULL;
  CALL_CUDA(hipMalloc(&ptr, size)) << " cuda malloc " << size << " failed.";
  LOG(INFO) << "Malloc " << ptr << " ... " << size;
  return ptr;
}

void CudaFree(void *ptr) {
  if (ptr) {
    CALL_CUDA(hipFree(ptr)) << " cuda free " << ptr << " failed.";
  }
  LOG(INFO) << "Free " << ptr;
}

void CudaMemcpyHostToDevice(void *dev, const void *host, size_t size) {
  CALL_CUDA(hipMemcpy(dev, host, size, hipMemcpyHostToDevice));
}

void CudaMemcpyDeviceToHost(void *host, const void *dev, size_t size) {
  CALL_CUDA(hipDeviceSynchronize());
  CALL_CUDA(hipMemcpy(host, dev, size, hipMemcpyDeviceToHost));
}
