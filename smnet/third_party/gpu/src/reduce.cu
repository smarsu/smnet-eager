#include "hip/hip_runtime.h"
// Copyright (c) 2020 smarsu. All Rights Reserved.

#include <vector>

#include "core.h"

struct ReduceParams {
  hipdnnTensorDescriptor_t x_desc;
  hipdnnTensorDescriptor_t y_desc;
  hipdnnReduceTensorDescriptor_t reduce_desc;

  size_t indice_size{0};
  size_t wksp_size{0};
  float alpha{1};
  float beta{0};

  bool need_indices{false};
  
  std::vector<int> x_strides;
  std::vector<int> y_strides;
};

extern "C" {

void DestroyReduceParams(ReduceParams *params) {
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->x_desc));
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->y_desc));
  CALL_CUDNN(hipdnnDestroyReduceTensorDescriptor(params->reduce_desc));

  delete params;
}

hipdnnReduceTensorOp_t GetReduceOp(int reduce_op) {
  static hipdnnReduceTensorOp_t ops[9] = {
    HIPDNN_REDUCE_TENSOR_ADD,
    HIPDNN_REDUCE_TENSOR_MUL,
    HIPDNN_REDUCE_TENSOR_MIN,
    HIPDNN_REDUCE_TENSOR_MAX,
    HIPDNN_REDUCE_TENSOR_AMAX,
    HIPDNN_REDUCE_TENSOR_AVG,
    HIPDNN_REDUCE_TENSOR_NORM1,
    HIPDNN_REDUCE_TENSOR_NORM2,
    HIPDNN_REDUCE_TENSOR_MUL_NO_ZEROS
  };

  CHECK_LT(reduce_op, 9);
  CHECK_GE(reduce_op, 0);

  return ops[reduce_op];
}

ReduceParams *CudnnReduceCreate(hipdnnHandle_t cudnn_handle,
                                int ndims,
                                int *x_dims,
                                int *y_dims,
                                int reduce_op,
                                bool need_indices) {
  ReduceParams *params = new ReduceParams;
  params->need_indices = need_indices;

  params->x_strides = Shape2Strides(x_dims, ndims);
  params->y_strides = Shape2Strides(y_dims, ndims);

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->x_desc));
  CALL_CUDNN(hipdnnSetTensorNdDescriptor(params->x_desc,
                                        HIPDNN_DATA_FLOAT,
                                        ndims,
                                        x_dims,
                                        params->x_strides.data()/* x_strides */));

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->y_desc));
  CALL_CUDNN(hipdnnSetTensorNdDescriptor(params->y_desc,
                                        HIPDNN_DATA_FLOAT,
                                        ndims,
                                        y_dims,
                                        params->y_strides.data()/* x_strides */));

  CALL_CUDNN(hipdnnCreateReduceTensorDescriptor(&params->reduce_desc));
  CALL_CUDNN(hipdnnSetReduceTensorDescriptor(params->reduce_desc,
                                            GetReduceOp(reduce_op),
                                            HIPDNN_DATA_FLOAT,
                                            HIPDNN_NOT_PROPAGATE_NAN,
                                            need_indices ? HIPDNN_REDUCE_TENSOR_FLATTENED_INDICES : HIPDNN_REDUCE_TENSOR_NO_INDICES,
                                            HIPDNN_32BIT_INDICES));

  CALL_CUDNN(cudnnGetReductionIndicesSize(cudnn_handle,
                                          params->reduce_desc,
                                          params->x_desc,
                                          params->y_desc,
                                          &params->indice_size));
  
  CALL_CUDNN(hipdnnGetReductionWorkspaceSize(cudnn_handle,
                                            params->reduce_desc,
                                            params->x_desc,
                                            params->y_desc,
                                            &params->wksp_size));

  size_t size = std::max(params->indice_size, params->wksp_size);
  CudaBuffer::Buffer()->Resize(size);

  LOG(INFO) << "Get Reduce Params ... " << size
            << " From " << ToString(x_dims, ndims) << " To " << ToString(y_dims, ndims);

  return params;
}

void CudnnReduceForward(hipdnnHandle_t cudnn_handle,
                        ReduceParams *params,
                        void *indices,
                        float alpha,
                        const void *x,
                        float beta,
                        void *y) {
  params->alpha = alpha;
  params->beta = beta;

  CALL_CUDNN(hipdnnReduceTensor(cudnn_handle,
                               params->reduce_desc,
                               indices,
                              //  params->need_indices ? indices : CudaBuffer::Buffer()->data(),
                               params->indice_size,
                               CudaBuffer::Buffer()->data(),
                               params->wksp_size,
                               &params->alpha,
                               params->x_desc,
                               x,
                               &params->beta,
                               params->y_desc,
                               y));

  CALL_CUDA(hipDeviceSynchronize());
}

}  // extern "C"
