// Copyright (c) 2020 smarsu. All Rights Reserved.

#include <vector>

#include "core.h"

struct ActivationParams {
  hipdnnTensorDescriptor_t x_desc;
  hipdnnTensorDescriptor_t y_desc;
  hipdnnActivationDescriptor_t act_desc;

  float fwd_alpha{1};
  float fwd_beta{0};
  float bwd_alpha{1};
  float bwd_beta{0};

  std::vector<int> strides;
};

extern "C" {

void DestroyActivationParams(ActivationParams *params) {
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->x_desc));
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->y_desc));
  CALL_CUDNN(hipdnnDestroyActivationDescriptor(params->act_desc));

  delete params;
}

hipdnnActivationMode_t GetActivationMode(int mode) {
  CHECK(mode >= 0 && mode < 5);

  static hipdnnActivationMode_t activation_modes[5] = {
    HIPDNN_ACTIVATION_SIGMOID,
    HIPDNN_ACTIVATION_RELU,
    HIPDNN_ACTIVATION_TANH,
    HIPDNN_ACTIVATION_CLIPPED_RELU,
    HIPDNN_ACTIVATION_ELU
  };

  return activation_modes[mode];
}

ActivationParams *CudnnActivationCreate(int ndims,
                                        int *shape,
                                        int mode,
                                        double coef) {
  ActivationParams *params = new ActivationParams;

  CHECK(ndims >= 4 && ndims <= CUDNN_DIM_MAX);

  params->strides = Shape2Strides(shape, ndims);

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->x_desc));
  CALL_CUDNN(hipdnnSetTensorNdDescriptor(params->x_desc,
                                        HIPDNN_DATA_FLOAT,
                                        ndims,
                                        shape,
                                        params->strides.data()/* x_strides */));

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->y_desc));
  CALL_CUDNN(hipdnnSetTensorNdDescriptor(params->y_desc,
                                        HIPDNN_DATA_FLOAT,
                                        ndims,
                                        shape,
                                        params->strides.data()/* x_strides */));

  CALL_CUDNN(hipdnnCreateActivationDescriptor(&params->act_desc));
  CALL_CUDNN(hipdnnSetActivationDescriptor(params->act_desc,
                                          GetActivationMode(mode),
                                          HIPDNN_NOT_PROPAGATE_NAN,
                                          coef));

  return params;
}

void CudnnActivationForward(hipdnnHandle_t cudnn_handle,
                            ActivationParams *params,
                            float alpha,
                            const void *x,
                            float beta,
                            void *y) {
  params->fwd_alpha = alpha;
  params->fwd_beta = beta;

  CALL_CUDNN(hipdnnActivationForward(cudnn_handle,
                                    params->act_desc,
                                    &params->fwd_alpha,
                                    params->x_desc,
                                    x,
                                    &params->fwd_beta,
                                    params->y_desc,
                                    y));
}

void CudnnActivationBackward(hipdnnHandle_t cudnn_handle,
                             ActivationParams *params,
                             float alpha,
                             const void *y,
                             const void *dy,
                             const void *x,
                             float beta,
                             void *dx) {
  params->bwd_alpha = alpha;
  params->bwd_beta = beta;

  CALL_CUDNN(hipdnnActivationBackward(cudnn_handle,
                                     params->act_desc,
                                     &params->bwd_alpha,
                                     params->y_desc,
                                     y,
                                     params->y_desc,
                                     dy,
                                     params->x_desc,
                                     x,
                                     &params->bwd_beta,
                                     params->x_desc,
                                     dx));
}

}  // extern "C"
