// Copyright (c) 2020 smarsu. All Rights Reserved.

#include "core.h"

struct SoftmaxParams {
  hipdnnTensorDescriptor_t x_desc;
  hipdnnTensorDescriptor_t y_desc;

  hipdnnSoftmaxAlgorithm_t algo;

  float fwd_alpha{1};
  float fwd_beta{0};

  float bwd_alpha{1};
  float bwd_beta{0};
};

extern "C" {

void DestroySoftmaxParams(SoftmaxParams *params) {
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->x_desc));
  CALL_CUDNN(hipdnnDestroyTensorDescriptor(params->y_desc));

  delete params;
}

hipdnnSoftmaxAlgorithm_t GetSoftmaxMode(int algo) {
  static hipdnnSoftmaxAlgorithm_t algos[2] = {
    HIPDNN_SOFTMAX_ACCURATE,
    HIPDNN_SOFTMAX_LOG
  };

  return algos[algo];
}

SoftmaxParams *CudnnSoftmaxCreate(int n,
                                  int c,
                                  int h,
                                  int algo) {
  SoftmaxParams *params = new SoftmaxParams;

  params->algo = GetSoftmaxMode(algo);

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->x_desc));
  CALL_CUDNN(hipdnnSetTensor4dDescriptor(params->x_desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n,
                                        c,
                                        h,
                                        1));

  CALL_CUDNN(hipdnnCreateTensorDescriptor(&params->y_desc));
  CALL_CUDNN(hipdnnSetTensor4dDescriptor(params->y_desc,
                                        HIPDNN_TENSOR_NCHW,
                                        HIPDNN_DATA_FLOAT,
                                        n,
                                        c,
                                        h,
                                        1));

  return params;
}

void CudnnSoftmaxForward(hipdnnHandle_t cudnn_handle,
                         SoftmaxParams *params,
                         float alpha,
                         const void *x,
                         float beta,
                         void *y) {
  params->fwd_alpha = alpha;
  params->fwd_beta = beta;

  CALL_CUDNN(hipdnnSoftmaxForward(cudnn_handle,
                                 params->algo,
                                 HIPDNN_SOFTMAX_MODE_CHANNEL,
                                 &params->fwd_alpha,
                                 params->x_desc,
                                 x,
                                 &params->fwd_beta,
                                 params->y_desc,
                                 y));
}

void CudnnSoftmaxBackward(hipdnnHandle_t cudnn_handle,
                          SoftmaxParams *params,
                          float alpha,
                          const void *y,
                          const void *dy,
                          float beta,
                          void *dx) {
  params->bwd_alpha = alpha;
  params->bwd_beta = beta;

  CALL_CUDNN(hipdnnSoftmaxBackward(cudnn_handle,
                                  params->algo,
                                  HIPDNN_SOFTMAX_MODE_CHANNEL,
                                  &params->bwd_alpha,
                                  params->y_desc,
                                  y,
                                  params->y_desc,
                                  dy,
                                  &params->bwd_beta,
                                  params->x_desc,
                                  dx));
}

}  // extern "C" 
